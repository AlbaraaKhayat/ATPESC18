#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void simpleKernel(int N, float *d_a, float *d_b){
	   
  // Convert thread and thread-block indices into array index 
  const int n  = threadIdx.x + blockDim.x*blockIdx.x;
	   
  // If index is in [0,N-1] add entries
  if(n<N)
    d_a[N-1-n] = d_b[n];
}

int main(int argc,char **argv){
  int N = 512; // size of array for this DEMO
     // HOST array
  float *h_a = (float*) calloc(N, sizeof(float));
  float *h_b = (float*) calloc(N, sizeof(float)); 
     // Allocate DEVICE array
  float *d_a, *d_b; 
  hipMalloc((void**) &d_a, N*sizeof(float));
  hipMalloc((void**) &d_b, N*sizeof(float));
  dim3 dimBlock(512,1,1);          // 512 threads per thread-block
  dim3 dimGrid((N+511)/512, 1, 1); // Enough thread-blocks to cover N
  
    // Init. HOST array    
  for(int n=0;n<N;++n){
      h_b[n] = n;
  }
  hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyDeviceToHost);    
  // Queue kernel on DEVICE
  simpleKernel <<< dimGrid, dimBlock >>> (N, d_a, d_b);
    


  // Transfer result from DEVICE to HOST
  hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
    
  // Print out result
  for(int n=0;n<N;++n) printf("h_a[%d] = %f\n", n, h_a[n]);
}
